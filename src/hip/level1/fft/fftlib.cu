#include "hip/hip_runtime.h"
#include "cudacommon.h"
#define _USE_MATH_DEFINES
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <hipfft.h>
#include "OptionParser.h"
#include "fftlib.h"

int fftDevice = -1;

bool do_dp;

//#define USE_CUFFT

#ifdef USE_CUFFT
hipfftHandle plan;
// Arrange blocks into 2D grid that fits into the GPU (for powers of two only)
inline dim3 grid2D(const int nblocks)
{
    int slices = 1;
    while (nblocks/slices > 65535)
    {
        slices *= 2;
    }
    return dim3(nblocks/slices, slices);
}

void printCUFFTError(const hipfftResult res)
{
    if (res != HIPFFT_SUCCESS)
    {
        cout << "CUFFT Error: ";
        if (res == HIPFFT_INVALID_PLAN)
        {
            cout << "Invalid Plan.\n";
        }
        else if (res == HIPFFT_INVALID_VALUE)
        {
            cout << "Invalid Value.\n";
        }
        else if (res == HIPFFT_INTERNAL_ERROR)
        {
            cout << "Internal Error .\n";
        }
        else if (res == HIPFFT_EXEC_FAILED)
        {
            cout << "FFT Exec failed.\n";
        }
        else if (res == HIPFFT_SETUP_FAILED)
        {
            cout << "Setup failed.\n";
        }
        else if (res == HIPFFT_UNALIGNED_DATA)
        {
            cout << "Unaligned data (unused).\n";
        }
    }
}

#else
#include "codelets.h"
#endif

template <class T2> __global__ void
chk512_device(const T2* __restrict__ work, const int half_n_cmplx,
    char* __restrict__ fail)
{
    int i, tid = threadIdx.x;
    T2 a[8], b[8];

    work += (blockIdx.y * gridDim.x + blockIdx.x) * 512 + tid;

    for (i = 0; i < 8; i++)
    {
        a[i] = work[i*64];
    }

    for (i = 0; i < 8; i++)
    {
        b[i] = work[half_n_cmplx+i*64];
    }

    for (i = 0; i < 8; i++)
    {
        if (a[i].x != b[i].x || a[i].y != b[i].y)
        {
            *fail = 1;
        }
    }
}


template <class T2> __global__ void
norm512_device(T2* __restrict__ work)
{
    int i, tid = threadIdx.x;

    work += (blockIdx.y * gridDim.x + blockIdx.x) * 512 + tid;

    for (i = 0; i < 8; i++)
    {
        work[i*64].x /= 512;
        work[i*64].y /= 512;
    }
}


void
init(OptionParser& op, const bool _do_dp, const int n_ffts)
{
    do_dp = _do_dp;
    if (fftDevice == -1)
    {
        if (op.getOptionVecInt("device").size() > 0)
        {
            fftDevice = op.getOptionVecInt("device")[0];
        }
        else
        {
            fftDevice = 0;
        }
        hipSetDevice(fftDevice);
        hipGetDevice(&fftDevice);
    }
#ifdef USE_CUFFT
    hipfftResult res;
    cerr << "init: initing plan, n_ffts=" << n_ffts << endl;
    if (do_dp)
    {
        res = hipfftPlan1d(&plan, 512, HIPFFT_Z2Z, n_ffts);
    }
    else
    {
        res = hipfftPlan1d(&plan, 512, HIPFFT_C2C, n_ffts);
    }
    if (res != HIPFFT_SUCCESS)
    {
        cout << "CUFFT Error in plan.\n";
    }
    else
    {
        cerr <<  "success...\n";
    }
#endif

}


void
forward(void* work, const int n_ffts)
{
#ifdef USE_CUFFT
    hipfftResult res;
    if (do_dp)
    {
        res = hipfftExecZ2Z(plan, (hipfftDoubleComplex*)work,
            (hipfftDoubleComplex*)work, HIPFFT_FORWARD);
    }
    else
    {
        res = hipfftExecC2C(plan, (hipfftComplex*)work,
            (hipfftComplex*)work, HIPFFT_FORWARD);
    }
    printCUFFTError(res);
    hipDeviceSynchronize();
    CHECK_CUDA_ERROR();
#else
    if (do_dp)
    {
        hipLaunchKernelGGL(HIP_KERNEL_NAME(FFT512_device<double2, double>), grid2D(n_ffts), 64, 0, 0, (double2*)work);
    }
    else
    {
        hipLaunchKernelGGL(HIP_KERNEL_NAME(FFT512_device<float2, float>), grid2D(n_ffts), 64, 0, 0, (float2*)work);
    }
    hipDeviceSynchronize();
    CHECK_CUDA_ERROR();
#endif
}


void
inverse(void* work, const int n_ffts)
{
#ifdef USE_CUFFT
    hipfftResult res;
    if (do_dp)
    {
        res = hipfftExecZ2Z(plan, (hipfftDoubleComplex*)work,
            (hipfftDoubleComplex*)work, HIPFFT_BACKWARD);
    }
    else
    {
        res = hipfftExecC2C(plan, (hipfftComplex*)work,
            (hipfftComplex*)work, HIPFFT_BACKWARD);
    }
    printCUFFTError(res);

    // normalize data...
    if (do_dp)
    {
        hipLaunchKernelGGL(HIP_KERNEL_NAME(norm512_device<double2>), grid2D(n_ffts), 64, 0, 0, (double2*)work);
    }
    else
    {
        hipLaunchKernelGGL(HIP_KERNEL_NAME(norm512_device<float2>), grid2D(n_ffts), 64, 0, 0, (float2*)work);
    }
    hipDeviceSynchronize();
    CHECK_CUDA_ERROR();
#else
    if (do_dp)
    {
        hipLaunchKernelGGL(HIP_KERNEL_NAME(IFFT512_device<double2, double>), grid2D(n_ffts), 64, 0, 0, (double2*)work);
    }
    else
    {
        hipLaunchKernelGGL(HIP_KERNEL_NAME(IFFT512_device<float2, float>), grid2D(n_ffts), 64, 0, 0, (float2*)work);
    }
    hipDeviceSynchronize();
    CHECK_CUDA_ERROR();
    // normalization built in to inverse...
#endif
}


int
check(void* work, void* check, const int half_n_ffts, const int half_n_cmplx)
{
    char result;

    if (do_dp)
    {
        hipLaunchKernelGGL(HIP_KERNEL_NAME(chk512_device<double2>), grid2D(half_n_ffts), 64, 0, 0, 
            (double2*)work, half_n_cmplx, (char*)check);
    }
    else
    {
        hipLaunchKernelGGL(HIP_KERNEL_NAME(chk512_device<float2>), grid2D(half_n_ffts), 64, 0, 0, 
            (float2*)work, half_n_cmplx, (char*)check);
    }
    hipMemcpy(&result, check, 1, hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR();

    return result;
}


void
allocHostBuffer(void** bufferp, unsigned long bytes)
{
    hipHostMalloc(bufferp, bytes);
    CHECK_CUDA_ERROR();
}

void
allocDeviceBuffer(void** bufferp, unsigned long bytes)
{
    hipMalloc(bufferp, bytes);
    CHECK_CUDA_ERROR();
}

void
freeHostBuffer(void* buffer)
{
    hipHostFree(buffer);
    CHECK_CUDA_ERROR();
}


void
freeDeviceBuffer(void* buffer)
{
    hipFree(buffer);
}

void
copyToDevice(void* to_device, const void* from_host,
    const unsigned long bytes)
{
    hipMemcpy(to_device, from_host, bytes, hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR();
}

void
copyFromDevice(void* to_host, const void* from_device,
    const unsigned long bytes)
{
    hipMemcpy(to_host, from_device, bytes, hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR();
}

